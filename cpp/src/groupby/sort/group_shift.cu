#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/gather.cuh>
#include <cudf/detail/groupby/group_shift.hpp>
#include <cudf/detail/groupby/sort_helper.hpp>
#include <cudf/detail/iterator.cuh>
#include <cudf/detail/scatter.hpp>
#include <cudf/scalar/scalar.hpp>
#include <cudf/types.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/iterator/transform_iterator.h>

namespace cudf {
namespace groupby {
namespace detail {

namespace {

constexpr size_type SAFE_GATHER_IDX = 0;

/**
 * @brief Functor to determine the location to set `fill_value` for groupby shift.
 */
template <bool ForwardShift, typename EdgeIterator>
struct group_shift_fill_functor {
  EdgeIterator group_edges_begin;
  size_type offset;
  size_type group_label, offset_to_edge;

  group_shift_fill_functor(EdgeIterator group_edges_begin, size_type offset)
    : group_edges_begin(group_edges_begin), offset(offset)
  {
  }

  __device__ size_type operator()(size_type i)
  {
    if (ForwardShift) {  // offset > 0
      group_label    = i / offset;
      offset_to_edge = i % offset;
    } else {  // offset < 0
      group_label    = -i / offset;
      offset_to_edge = i % offset + offset + 1;
    }
    return *(group_edges_begin + group_label) + offset_to_edge;
  }
};

}  // namespace

/**
 * @brief Implementation of groupby shift
 *
 * Groupby shift is based on sort groupby. The first step is a global shift for `sorted_values`.
 * The second step is to set the proper locations to `fill_values`.
 *
 * @tparam EdgeIterator Iterator type to the group edge list
 *
 * @param sorted_values values to be sorted, grouped by keys
 * @param offset The off set by which to shift the input
 * @param fill_value Fill value for indeterminable outputs
 * @param group_bound_begin Beginning of iterator range of the list that contains indices to the
 * group's boundary. For forward shifts, the indices point to the groups' left boundaries, and right
 * boundaries otherwise
 * @param num_groups The number of groups
 * @param mr Device memory resource used to allocate the returned table's device memory
 * @param stream CUDA stream used for device memory operations and kernel launches
 * @return Column where values are shifted each group
 */
template <bool ForwardShift, typename EdgeIterator>
std::unique_ptr<column> group_shift_impl(column_view const& sorted_values,
                                         size_type offset,
                                         cudf::scalar const& fill_value,
                                         EdgeIterator group_bound_begin,
                                         std::size_t num_groups,
                                         rmm::cuda_stream_view stream,
                                         rmm::mr::device_memory_resource* mr)
{
  // Step 1: global shift
  auto shift_func = [col_size = sorted_values.size(), offset] __device__(size_type idx) {
    auto raw_shifted_idx = idx - offset;
    return static_cast<uint32_t>(
      raw_shifted_idx >= 0 and raw_shifted_idx < col_size ? raw_shifted_idx : SAFE_GATHER_IDX);
  };
  auto gather_iter_begin = cudf::detail::make_counting_transform_iterator(0, shift_func);

  auto shifted = cudf::detail::gather(table_view({sorted_values}),
                                      gather_iter_begin,
                                      gather_iter_begin + sorted_values.size(),
                                      out_of_bounds_policy::DONT_CHECK,
                                      stream,
                                      mr);

  // Step 2: set `fill_value`
  auto scatter_map = make_numeric_column(
    data_type(type_id::UINT32), num_groups * std::abs(offset), mask_state::UNALLOCATED);
  group_shift_fill_functor<ForwardShift, decltype(group_bound_begin)> fill_func{group_bound_begin,
                                                                                offset};
  auto scatter_map_iterator = cudf::detail::make_counting_transform_iterator(0, fill_func);
  thrust::copy(rmm::exec_policy(stream),
               scatter_map_iterator,
               scatter_map_iterator + scatter_map->view().size(),
               scatter_map->mutable_view().begin<size_type>());

  auto shifted_filled =
    cudf::detail::scatter({fill_value}, scatter_map->view(), shifted->view(), true, stream, mr);

  return std::move(shifted_filled->release()[0]);
}

std::unique_ptr<column> group_shift(column_view const& sorted_values,
                                    size_type offset,
                                    scalar const& fill_value,
                                    device_span<size_type const> group_offsets,
                                    rmm::cuda_stream_view stream,
                                    rmm::mr::device_memory_resource* mr)
{
  if (sorted_values.empty()) { return make_empty_column(sorted_values.type()); }

  if (offset > 0) {
    return group_shift_impl<true>(sorted_values,
                                  offset,
                                  fill_value,
                                  group_offsets.begin(),
                                  group_offsets.size() - 1,
                                  stream,
                                  mr);
  } else {
    auto rbound_iter = thrust::make_transform_iterator(group_offsets.begin() + 1,
                                                       [] __device__(auto i) { return i - 1; });
    return group_shift_impl<false>(
      sorted_values, offset, fill_value, rbound_iter, group_offsets.size() - 1, stream, mr);
  }
}

}  // namespace detail
}  // namespace groupby
}  // namespace cudf
