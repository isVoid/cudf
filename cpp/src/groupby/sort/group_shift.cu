#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include <cudf/utilities/span.hpp>
 #include <cudf/types.hpp>
 #include <cudf/scalar/scalar.hpp>
 #include <cudf/copying.hpp>
 #include <cudf/detail/iterator.cuh>
 #include <cudf/column/column_factories.hpp>
 #include <cudf/utilities/span.hpp>
 #include <cudf/detail/groupby/group_shift.hpp>
 #include <cudf/detail/gather.cuh>
 #include <cudf/detail/scatter.hpp>
 #include <cudf/detail/groupby/sort_helper.hpp>
 
 #include <cudf/debug_printers.hpp>

 #include <rmm/cuda_stream_view.hpp>
 #include <rmm/exec_policy.hpp>
 #include <rmm/device_uvector.hpp>

 #include <thrust/binary_search.h>

#include <cxxabi.h>

 namespace cudf {
 namespace groupby {
 namespace detail {

namespace {

constexpr size_type SAFE_GATHER_IDX = 0;

// template<bool ForwardShift = true>
// bool is_valid_shifted_element(cudf::device_span<size_type const> group_offsets, size_type const& offset, rmm::cuda_stream_view stream) {
//     auto base_group_offset = *(thrust::upper_bound(rmm::exec_policy(stream), group_offsets.begin(), group_offsets.end(), idx) - 1);
//     if (ForwardShift)
//         return (i - base_group_offset) >= offset;
//     else
//         return (i - base_group_offset) > offset;
// }

template<bool ForwardShift, typename EdgeIterator>
struct group_shift_fill_functor {
    EdgeIterator group_edges_begin;
    size_type offset;
    size_type group_label, offset_to_edge;
    // template<bool ForwardShift = true>
    // bool __device__ operator()(size_type i) {
    //     auto base_group_offset = *(thrust::upper_bound(rmm::exec_policy(stream), group_offsets.begin(), group_offsets.end(), idx) - 1);
    //     if (ForwardShift)
    //         return (i - base_group_offset) >= offset;
    //     else
    //         return (i - base_group_offset) > offset;
    // }

    group_shift_fill_functor(EdgeIterator group_edges_begin, size_type offset) : group_edges_begin(group_edges_begin), offset(offset) {
    }

    __device__ size_type operator()(size_type i) {
        if (ForwardShift) { // offset > 0
            group_label = i / offset;
            offset_to_edge = i % offset;
        }
        else { // offset < 0
            group_label = -i / offset;
            offset_to_edge = -i % offset + offset + 1;
        }
        return *(group_edges_begin + group_label) + offset_to_edge;
    }

};

}   // namespace anonymous

template<bool ForwardShift, typename EdgeIterator>
 std::unique_ptr<column> group_shift_impl(column_view const& values,
                                     size_type offset,
                                     EdgeIterator group_edges_begin,
                                     std::size_t num_groups,
                                     cudf::scalar const& fill_value,
                                     rmm::cuda_stream_view stream,
                                     rmm::mr::device_memory_resource* mr)
 {
    auto shift_func = [col_size = values.size(), offset] __device__ (size_type idx) {
       auto raw_shifted_idx = idx - offset;
       return static_cast<uint32_t>(raw_shifted_idx >= 0 and raw_shifted_idx < col_size ? raw_shifted_idx : SAFE_GATHER_IDX);
   };
   auto gather_iter_begin = cudf::detail::make_counting_transform_iterator(0, shift_func);

   auto shifted = cudf::detail::gather(table_view({values}), gather_iter_begin, gather_iter_begin + values.size(), out_of_bounds_policy::DONT_CHECK, stream, mr);

   auto scatter_map = make_numeric_column(data_type(type_id::UINT32), num_groups * std::abs(offset), mask_state::UNALLOCATED);
   group_shift_fill_functor<ForwardShift, decltype(group_edges_begin)> fill_func{group_edges_begin, offset};
   
   if (ForwardShift) {
        auto scatter_map_iterator = cudf::detail::make_counting_transform_iterator(0, fill_func);
        thrust::copy(rmm::exec_policy(stream), scatter_map_iterator, scatter_map_iterator + scatter_map->view().size(), scatter_map->mutable_view().begin<size_type>());
   }
//    else {
//        rmm::device_uvector<size_type> group_edges(group_offsets.size(), stream);
//        auto binop = [] __device__ (size_type const& grp_off, size_type const &grp_sz){
//            return grp_off + grp_sz - 1;
//        };
//        thrust::transform(group_offsets.begin(), group_offsets.end(), group_sizes.begin(), group_edges.begin(), binop);
//        group_shift_fill_functor<false> fill_func{group_edges, offset};
//        auto scatter_map_iterator = cudf::detail::make_counting_transform_iterator(0, fill_func);
//        thrust::copy(scatter_map_iterator, scatter_map_iterator + scatter_map->view().size(), scatter_map->mutable_view().begin<size_type>());
//    }

// std::reference_wrapper<const cudf::scalar> slr_ref{fill_value};
// std::vector<std::reference_wrapper<const cudf::scalar>> slr_vec{slr_ref};
   auto shifted_filled = cudf::detail::scatter({fill_value}, scatter_map->view(), shifted->view(), true, stream, mr);

   return std::move(shifted_filled->release()[0]);
 }

 std::unique_ptr<column> group_shift(
    column_view const& values,
    size_type offset,
    scalar const& fill_value,
    sort::sort_groupby_helper &helper,
    rmm::cuda_stream_view stream,
    rmm::mr::device_memory_resource* mr) 
  {
    if (values.size() == 0) {
      return make_empty_column(values.type());
    }
  
    rmm::device_uvector<size_type> const& group_offsets = helper.group_offsets(stream);
    if (offset > 0) {
        return group_shift_impl<true>(values, offset, group_offsets.begin(), group_offsets.size() - 1, fill_value, stream, mr);
    }
    // else {
    //     rmm::device_uvector<size_type> group_sizes(group_offsets.size()-1, stream);
    //     thrust::transform(rmm::exec_policy(stream),
    //                       group_offsets.begin(),
    //                       group_offsets.begin() + group_sizes.size(),
    //                       group_offsets.begin() + 1,
    //                       group_sizes.begin(),
    //                       [] __device__ (auto offset_cur, auto offset_next) {return offset_next - offset_cur;});
    //     return group_shift_impl<false>(values, offset, group_offsets, fill_value, stream, mr);
    // }
    return make_numeric_column(data_type(type_id::INT32), 1, mask_state::UNALLOCATED);
  }

}  // namespace detail
}  // namespace groupby
}  // namespace cudf
